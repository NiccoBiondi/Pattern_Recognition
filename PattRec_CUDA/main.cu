//
// Created by nicco on 04/03/20.
//

#include <iostream>
#include <chrono>
#include <vector>
#include <sstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand.h>
#include <thrust/sort.h>

#include "execution.cuh"
#include "iteration.cuh"
#include "macros.h"

__constant__ float queries_const[MAX_LEN_Q];

int main(int argc, char **argv) {

#ifdef __HIPCC__
    std::cout << "cuda defined" << std::endl;
#endif

    std::cout << "Welcome to Pattern Recognition" << std::endl;

    // default hyper-parameters
    int LEN_SEQ = 10;
    int LEN_PATTERN_SEQ = 6;
    int NUM_QUERIES = 2;
    int verbose = 1;
    int iterations = 2;
    std::string type = "n";
    std::string mode = "tiling";     // mode: naive private tiling or constant
    // number of runs to compute speed up mean and std
    int RUNS = 2;

    // set other hyper-parameters with launch arguments
    if (argc == 8) {
        // convert the string argv[1] parameter in int
        std::string s_LEN_SEQ = argv[1];
        std::stringstream parser1(s_LEN_SEQ);
        parser1 >> LEN_SEQ;

        std::string s_LEN_PATTERN_SEQ = argv[2];
        std::stringstream parser2(s_LEN_PATTERN_SEQ);
        parser2 >> LEN_PATTERN_SEQ;

        std::string s_NUM_QUERIES = argv[3];
        std::stringstream parser3(s_NUM_QUERIES);
        parser3 >> NUM_QUERIES;

        std::string s_runs = argv[4];
        std::stringstream parser4(s_runs);
        parser4 >> RUNS;

        std::string s_iter = argv[5];
        std::stringstream parser5(s_iter);
        parser5 >> iterations;

        type = argv[6];

        std::string s_verbose = argv[7];
        std::stringstream parser6(s_verbose);
        parser6 >> verbose;

        if (LEN_SEQ < LEN_PATTERN_SEQ) {
            std::cout << "len of historical data less than len pattern seq!! Try again! " << std::endl;
            return 1;
        }

        std::cout << "You choose the following hyper-parameters: \n" << RUNS << " number of runs for mean and std; "
                  << type << " as type of execution\n " << NUM_QUERIES << " as number of queries; " << LEN_SEQ
                  << " as len of historical data; "
                  << LEN_PATTERN_SEQ << " as len of each query; " << verbose << " as verbose." << std::endl;
    }

    float *statistic;
    int size = iterations * 3;
    statistic = (float *) malloc(size * sizeof(float));

    for (int it = 0; it < iterations*3; it=it+3) {
        one_iteration(LEN_SEQ, LEN_PATTERN_SEQ, NUM_QUERIES, RUNS, type, mode, verbose, statistic, it);
        LEN_SEQ *= 5;
    }

    // save in csv statistics
    save_result(statistic, size, mode);

    return 0;
}